#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <cuda/std/limits>
#include <numeric>
#include <optional>
#include <sstream>
#include <vector>

#include "../support/logging.h"
#include "kernels.h"

#define XGRAMMAR_CUDA_CALL(...)                                                                    \
  do {                                                                                             \
    __VA_ARGS__;                                                                                   \
    hipError_t err = hipGetLastError();                                                          \
    XGRAMMAR_CHECK(err == hipSuccess) << "CUDA Error: " << hipGetErrorString(err) << " (" << err \
                                       << ") " << __FILE__ << ": line " << __LINE__ << std::endl;  \
  } while (0)

#define XGRAMMAR_DISPATCH_DTYPE(dtype_flag, c_type, ...)                                         \
  do {                                                                                           \
    switch (dtype_flag) {                                                                        \
      case DTypeFlag::DTYPE_FLOAT16: {                                                           \
        using c_type = half;                                                                     \
        __VA_ARGS__;                                                                             \
        break;                                                                                   \
      }                                                                                          \
      case DTypeFlag::DTYPE_FLOAT32: {                                                           \
        using c_type = float;                                                                    \
        __VA_ARGS__;                                                                             \
        break;                                                                                   \
      }                                                                                          \
      case DTypeFlag::DTYPE_FLOAT64: {                                                           \
        using c_type = double;                                                                   \
        __VA_ARGS__;                                                                             \
        break;                                                                                   \
      }                                                                                          \
      default:                                                                                   \
        std::ostringstream oss;                                                                  \
        oss << #__VA_ARGS__ << " failed to dispatch data type " << static_cast<int>(dtype_flag); \
        XGRAMMAR_LOG(FATAL) << oss.str();                                                        \
        break;                                                                                   \
    }                                                                                            \
  } while (0)

namespace xgrammar {

#define BITS_PER_BLOCK 32
#define THREADS_PER_BLOCK 1024
#define ELEMENTS_PER_THREAD 4
#define CEIL_DIV(a, b) (((a) + (b) - 1) / (b))
#define GET_BIT(data_ptr, bit_idx) \
  ((data_ptr[(bit_idx) / BITS_PER_BLOCK] >> ((bit_idx) % BITS_PER_BLOCK)) & 1)

template <typename T>
__device__ T GetNegativeInfinity() {
  return -cuda::std::numeric_limits<T>::infinity();
}

template <>
__device__ half GetNegativeInfinity<half>() {
  return __float2half(-INFINITY);
}

__global__ void __launch_bounds__(1024) ApplyTokenBitmaskInplaceKernel(
    float* __restrict__ logits,
    const int32_t* __restrict__ bitmask,
    const int32_t* __restrict__ indices,
    int vocab_size,
    int bitmask_size
) {
  int bid = indices[blockIdx.y];
  // printf("accessing by: %d, bid: %d\n", blockIdx.y, bid);
  int tid = (blockIdx.x * blockDim.x + threadIdx.x) * ELEMENTS_PER_THREAD;

  float* logits_ptr = logits + bid * vocab_size + tid;

  for (int i = 0; i < ELEMENTS_PER_THREAD && tid + i < vocab_size; ++i) {
    // logits[bid, tid + i] = mask(..., bitmask[by, tid + i])
    if (GET_BIT(reinterpret_cast<const int32_t*>(bitmask + blockIdx.y * bitmask_size), tid + i) ==
        0) {
      logits_ptr[i] = GetNegativeInfinity<float>();
    }
  }
}

void ApplyTokenBitmaskInplace(
    float* logits,
    int32_t* bitmask,
    int batch_size,
    int vocab_size,
    std::optional<std::vector<int>> indices
) {
  if (indices) {
    for (int i = 0; i < indices->size(); ++i) {
      XGRAMMAR_CHECK(indices->at(i) < batch_size)
          << "index " << indices->at(i) << " is out of bounds";
    }
  } else {
    indices = std::vector<int>(batch_size);
    std::iota(indices->begin(), indices->end(), 0);
  }

  dim3 num_blocks(CEIL_DIV(vocab_size, THREADS_PER_BLOCK * ELEMENTS_PER_THREAD), indices->size());
  int num_threads = THREADS_PER_BLOCK;

  int* device_indices;
  hipMalloc(&device_indices, indices->size() * sizeof(int));
  hipMemcpy(
      device_indices, indices->data(), indices->size() * sizeof(int), hipMemcpyHostToDevice
  );

  XGRAMMAR_CUDA_CALL(ApplyTokenBitmaskInplaceKernel<<<num_blocks, num_threads>>>(
      logits, bitmask, device_indices, vocab_size, CEIL_DIV(vocab_size, BITS_PER_BLOCK)
  ));
}

}  // namespace xgrammar
